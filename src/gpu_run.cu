#include "hip/hip_runtime.h"
#include "gpu_device_interaction.cuh"
#include "gpu_run.cuh"
#include "gpu_ray_kernel.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_geometry_operations.cuh"

#include "patient_parameters.hpp"
#include "special_types.hpp"
#include "tramp.hpp"
#include "spot.hpp"

#include <iostream>
#include <fstream>
#include <vector>
#include <sys/stat.h>

void init_rays(const Patient_Parameters_t& pat,
               const unsigned int ibeam,
               std::vector<float4>& xbuffer,
               std::vector<float4>& vxbuffer)
//  initialize particle buffer
{
    std::cout << "Loading beam: " << pat.beam_names.at(ibeam) << std::endl;
    std::cout << "    Beam number:  " << ibeam << std::endl;
    std::cout << "    Machine:      " << pat.machine << std::endl;
    std::cout << "    Tramp file:   " << pat.tramp_files.at(ibeam) << std::endl;

    Tramp_t src(pat.tramp_files.at(ibeam), pat.machine);
    src.z = pat.isocenter_to_beam_distance.at(ibeam); // cm
    src.zeff = src.z;

    if(pat.range_shifters[ibeam].exists)
    {
        RangeShifter_Dims_t rs = pat.range_shifters[ibeam];
        std::cout << "    Range shifter thickness:    " << rs.thick << " cm" << std::endl;
        for (size_t i = 0; i < src.wepls.size(); i++)
        {
            src.wepls.at(i) -= rs.wepl;
        }
    }
    if(pat.apertures[ibeam].exists)
    {
        Aperture_Dims_t ap = pat.apertures[ibeam];
        std::cout << "    Aperture thickness:    " << ap.thick << " cm" << std::endl;
        std::cout << "    Aperture z downstream: " << ap.zdown << " cm" << std::endl;
    }
    std::cout << "    Source Z plane: " << src.z << " cm" << std::endl;

    float2 SAD = make_float2(pat.virtualSAD.a, pat.virtualSAD.b); // cm

    xbuffer.reserve(src.nspots);
    vxbuffer.reserve(src.nspots);
    for(unsigned int i=0; i < src.nspots; i++)
    { // LOOP OVER SPOTS
        Spot_t& spot = src.spots[i];
        // From fluence map coordinates to MCAuto coordinates.
        // This is the coordinate system in which the phase space files are written.
        // Then I just have to perform the same transformations as with the phase spaces.
        float3 pos; // cm
        // The x,y coordinates are push together or pulled appart depending on the z coord they are going to be initialized in.
        // Some angular spread will be added too
        pos.x = ((SAD.x + src.z) / SAD.x) * spot.x;
        pos.y = ((SAD.y + src.z) / SAD.y) * spot.y;
        pos.z = src.z;

        float3 dCos;
        float a = (spot.x-pos.x)/abs(src.z);
        float b = (spot.y-pos.y)/abs(src.z);
        float norm = sqrt(a*a + b*b + 1.f);
        dCos.x = a/norm;
        dCos.y = b/norm;

        float temp = 1.0f - dCos.x*dCos.x - dCos.y*dCos.y;
        if(temp < 0)
        {
            std::cerr << "Something went wrong calculating direction cosines:" << std::endl;
            std::cerr << "Are these correct and they make sense?" << std::endl;
            std::cerr << "    Nominal Z plane:   " << src.z       << std::endl;
            std::cerr << "    Effective Z plane: " << src.z    << std::endl;
            std::cerr << "    SAD.x:             " << SAD.x       << std::endl;
            std::cerr << "    SAD.y:             " << SAD.y       << std::endl;
            std::cerr << "    spot #:            " << i           << std::endl;
            std::cerr << "    spot x:            " << spot.x      << std::endl;
            std::cerr << "    spot y:            " << spot.y      << std::endl;
            exit(EXIT_FAILURE);
        };
        dCos.z = sqrt(temp);

        // From energy to WEPL
        float energy = src.energies_internal.at(i)*1000000; // eV
        float wepl   = src.wepls.at(i);

        xbuffer.push_back( make_float4( -pos.y, -pos.x, pos.z, wepl) );
        vxbuffer.push_back( make_float4( -dCos.y, -dCos.x, dCos.z, energy) );
    }
}


unsigned int rays_to_device(const std::vector<float4>& xbuffer,
                            const std::vector<float4>& vxbuffer,
                            const float2& angles,
                            const float3& ct_offsets)
{
    unsigned int num = xbuffer.size();

    // prepare GPU
    size_t bytes = sizeof(float4)*num;
    gpuErrchk( hipMalloc((void **) &xdata, bytes) );
    gpuErrchk( hipMalloc((void **) &vxdata, bytes) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(xdata),  xbuffer.data(), bytes, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(vxdata), vxbuffer.data(), bytes, 0, hipMemcpyHostToDevice) );

    int nblocks = 1 + (num-1)/NTHREAD_PER_BLOCK_SOURCE;
    rays_to_device_kernel<<<nblocks, NTHREAD_PER_BLOCK_SOURCE>>>(num, angles, ct_offsets);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    return num;
}


void calculateRays(const std::vector<float4>& xbuffer,
                   const std::vector<float4>& vxbuffer,
                   const BeamAngles_t& ang,
                   const float3& ct_offsets)
{
    float2 angles = make_float2(ang.gantry, ang.couch);

    unsigned int num = rays_to_device(xbuffer, vxbuffer, angles, ct_offsets);

    //      simulate a batch of rays
    std::cout << "\tCalculating " << num << " rays ..." << std::endl;
    int nblocks = 1 + (num-1)/NTHREAD_PER_BLOCK_RAYS;
    calculateRays_kernel<<<nblocks, NTHREAD_PER_BLOCK_RAYS>>>(num, scorer);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}


void outputScorerResults(float* src, const size_t size, std::string beam_name, std::string dir)
{
    std::cout << "Output results" << std::endl;
    std::string file = dir + "/" + beam_name + ".bin";
    std::cout << "\t" << file << std::endl;

    struct stat sb;
    if (!(stat(dir.c_str(), &sb) == 0 && S_ISDIR(sb.st_mode)))
        mkdir(dir.c_str(), 0774);        

    // copy data from GPU to CPU
    std::vector<float> temp(size);
    gpuErrchk( hipMemcpy(temp.data(), src, size, hipMemcpyDeviceToHost) );

    // write results to file
    std::ofstream ofs(file, std::ofstream::out | std::ofstream::binary);
    if(!ofs.is_open())
    {
        std::cerr << "Can't open file " << file << " to write results." << std::endl;
        exit(EXIT_FAILURE);
    }
    ofs.write(reinterpret_cast<const char*>(temp.data()), size);
}


void setScorerToZeros(void *s, size_t sz)
{
    hipMemset(s, 0, sz);
}

void freeMemory()
{ 
    hipFreeArray(dens);
    hipUnbindTexture(dens_tex);
    hipFree(scorer);
}


