#include "hip/hip_runtime.h"
#include "gpu_device_globals.cuh"

__device__ double4 xdata[NRAYS];  // x y z wepl (in cm)
__device__ double4 vxdata[NRAYS]; // vx vy vz (normalized) energy (eV)
__device__ short2 ixdata[NRAYS]; // particle metadata (beam and spot ID)

// CT variables
__device__ __constant__ double3 ctVoxSize;
__device__ __constant__ int3 ctVox;
__device__ __constant__ size_t ctTotalVoxN;

// Density-filled array
hipArray* dens;
texture<float, 3, hipReadModeElementType> dens_tex;

// Material ID-filled array
hipArray* matid;
texture<float, 3, hipReadModeElementType> matid_tex;

// Water resticted stopping power
__device__ __constant__ float stp_w_min_e;
__device__ __constant__ float stp_w_delta_e;
hipArray *stp_w_array, *stp_w_b_coeff_array;
texture<float,1,hipReadModeElementType> stp_w_tex, stp_w_b_coeff_tex;

// Stopping power ratio table
__device__ __constant__ float stp_ratio_min_e;
__device__ __constant__ float stp_ratio_delta_e;
hipArray* stp_ratio_array;
texture<float, 2, hipReadModeElementType> stp_ratio_tex;

// Bragg peaks LUTs
__device__ __constant__ float bp_energy_min;
__device__ __constant__ float bp_energy_delta;
__device__ __constant__ float bp_depth_delta;
hipArray* bp_n_array;
texture<float, 2, hipReadModeElementType> bp_n_tex;
hipArray* bp_w_array;
texture<float, 2, hipReadModeElementType> bp_w_tex;
hipArray* bp_s_array;
texture<float, 2, hipReadModeElementType> bp_s_tex;
hipArray* bp_b_array;
texture<float, 2, hipReadModeElementType> bp_b_tex;