#include "hip/hip_runtime.h"
#include "gpu_device_globals.cuh"

__device__ float4 xdata[NRAYS];  // x y z wepl (in cm)
__device__ float4 vxdata[NRAYS]; // vx vy vz (normalized) energy (eV)
__device__ short2 ixdata[NRAYS]; // particle metadata (beam and spot ID)

// CT variables
__device__ __constant__ float3 ctVoxSize;
__device__ __constant__ int3 ctVox;
__device__ __constant__ size_t ctTotalVoxN;

// Density-filled array
hipArray* dens;
texture<float, hipTextureType3D, hipReadModeElementType> dens_tex;

// Material ID-filled array
hipArray* matid;
texture<float, hipTextureType3D, hipReadModeElementType> matid_tex;

// Boolean mask array
__device__ bool masking_vf;
hipArray* vf_mask;
texture<int, hipTextureType3D, hipReadModeElementType> vf_mask_tex;

// Water resticted stopping power
__device__ __constant__ float stp_w_min_e;
__device__ __constant__ float stp_w_delta_e;
hipArray *stp_w_array, *stp_w_b_coeff_array;
texture<float, hipTextureType1D, hipReadModeElementType> stp_w_tex, stp_w_b_coeff_tex;

// Stopping power ratio table
__device__ __constant__ float stp_ratio_min_e;
__device__ __constant__ float stp_ratio_delta_e;
hipArray* stp_ratio_array;
texture<float, hipTextureType2D, hipReadModeElementType> stp_ratio_tex;

// Bragg peaks LUTs
__device__ __constant__ float bp_energy_min;
__device__ __constant__ float bp_energy_delta;
__device__ __constant__ float bp_depth_delta;
hipArray* bp_n_array;
texture<float, hipTextureType2D, hipReadModeElementType> bp_n_tex;
hipArray* bp_w_array;
texture<float, hipTextureType2D, hipReadModeElementType> bp_w_tex;
hipArray* bp_s_array;
texture<float, hipTextureType2D, hipReadModeElementType> bp_s_tex;
hipArray* bp_b_array;
texture<float, hipTextureType2D, hipReadModeElementType> bp_b_tex;
hipArray* bp_range_array;
texture<float, hipTextureType1D, hipReadModeElementType> bp_range_tex;
