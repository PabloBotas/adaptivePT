#include "hip/hip_runtime.h"
#include "gpu_utils.cuh"

#include "gpu_errorcheck.cuh"
#include "vector4.hpp"

///////////////////////////////////////
template <class T>
void allocate_scorer(T*& s, size_t n)
{
    gpuErrchk( hipMalloc( (void **) &s, sizeof(T)*n) );
    gpuErrchk( hipMemset( (void *) s, 0, sizeof(T)*n) );
}
template void allocate_scorer<float4>(float4*&, size_t);
template void allocate_scorer<short2>(short2*&, size_t);
template void allocate_scorer<float>(float*&, size_t);

///////////////////////////////////////
template <class S, class T>
void retrieve_scorer(S* h, T* d, size_t n)
{
    gpuErrchk( hipMemcpy(h, d, sizeof(T)*n, hipMemcpyDeviceToHost) );
}
template void retrieve_scorer<short, short2>(short*, short2*, size_t);
template void retrieve_scorer<float, float>(float*, float*, size_t);
template void retrieve_scorer<float, float4>(float*, float4*, size_t);

///////////////////////////////////////
template <class T>
void array_to_device(T*& dest, const T* src, size_t n)
{
    gpuErrchk( hipMalloc((void **) &dest, sizeof(T)*n) );
    gpuErrchk( hipMemcpy(dest, src, sizeof(T)*n, hipMemcpyHostToDevice) );
}
template void array_to_device<short>(short*&, const short*, size_t);
template void array_to_device<float2>(float2*&, const float2*, size_t);

///////////////////////////////////////

// template <class T>
// void symbol_to_device(T*& dest, const T* src, size_t n)
// {
//     gpuErrchk( hipMalloc((void **) &dest, sizeof(T)*n) );
//     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dest),  src, sizeof(T)*n, 0, hipMemcpyHostToDevice) );
// }
// template void symbol_to_device<float4>(float4*& dest, const float4*, size_t);
// template void symbol_to_device<short2>(short2*& dest, const short2*, size_t);

///////////////////////////////////////
template <class T>
void sendVectorToTexture(size_t w, size_t h, size_t d,
                         std::vector<T> host_vec,
                         hipArray* array,
                         texture<T, 3, hipReadModeElementType>& tex)
{
    //  create a 3d array on device
    hipExtent extent = make_hipExtent(w, h, d);
    gpuErrchk( hipMalloc3DArray(&array, &tex.channelDesc, extent) );

    // copy data to GPU
    hipMemcpy3DParms pars = {0};
    pars.srcPtr   = make_hipPitchedPtr((void *)host_vec.data(),
                                        extent.width*sizeof(T),
                                        extent.width, extent.height);
    pars.dstArray = array;
    pars.extent   = extent;
    pars.kind     = hipMemcpyHostToDevice;
    gpuErrchk( hipMemcpy3D(&pars) );
    // Bind device array to texture
    tex.normalized = false;
    tex.filterMode = hipFilterModePoint;
    gpuErrchk( hipBindTextureToArray(tex, array, tex.channelDesc) );
}
template void sendVectorToTexture<float>(size_t w, size_t h, size_t d,
                                         std::vector<float> host_vec,
                                         hipArray* array,
                                         texture<float, 3, hipReadModeElementType>& tex);

