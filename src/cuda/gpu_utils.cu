#include "hip/hip_runtime.h"
#include "gpu_utils.cuh"

#include "gpu_errorcheck.cuh"

///////////////////////////////////////
template <class T>
void allocate_scorer(T*& s, size_t n)
{
    gpuErrchk( hipMalloc( (void **) &s, sizeof(T)*n) );
    gpuErrchk( hipMemset( (void *) s, 0, sizeof(T)*n) );
}
template void allocate_scorer<float4>(float4*&, size_t);
template void allocate_scorer<short2>(short2*&, size_t);
template void allocate_scorer<float>(float*&, size_t);

///////////////////////////////////////
template <class S, class T>
void retrieve_scorer(S* h, T* d, size_t n)
{
    gpuErrchk( hipMemcpy(h, d, sizeof(T)*n, hipMemcpyDeviceToHost) );
}
template void retrieve_scorer<float, float4>(float*, float4*, size_t);

///////////////////////////////////////
template <class T>
void array_to_device(T*& dest, const T* src, size_t n)
{
    gpuErrchk( hipMalloc((void **) &dest, sizeof(T)*n) );
    gpuErrchk( hipMemcpy(dest, src, sizeof(T)*n, hipMemcpyHostToDevice) );
}
template void array_to_device<short>(short*&, const short*, size_t);
template void array_to_device<float2>(float2*&, const float2*, size_t);

///////////////////////////////////////

// template <class T>
// void symbol_to_device(T*& dest, const T* src, size_t n)
// {
//     gpuErrchk( hipMalloc((void **) &dest, sizeof(T)*n) );
//     gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dest),  src, sizeof(T)*n, 0, hipMemcpyHostToDevice) );
// }
// template void symbol_to_device<float4>(float4*& dest, const float4*, size_t);
// template void symbol_to_device<short2>(short2*& dest, const short2*, size_t);
