#include "gpu_errorcheck.cuh"
#include "cuda_helper/hip/hip_runtime_api.h"

#include <iostream>
#include <string>

#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__); }

void gpuAssert(hipError_t code, char const *file, int line, bool abort)
{
    if (code != hipSuccess) {
        std::cerr << "Error code " << code << ": " << hipGetErrorString(code) << std::endl;
        std::cerr << "File: " << file << std::endl;
        std::cerr << "Line: " << line << std::endl;
        if (abort)
            exit(code);
    }
}

void check_kernel_execution(char const *file, int line)
{
    hipError_t code = hipDeviceSynchronize();
    gpuAssert(code, file, line);
    code = hipPeekAtLastError();
    gpuAssert(code, file, line);
}
