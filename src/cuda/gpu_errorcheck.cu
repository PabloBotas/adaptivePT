#include "gpu_errorcheck.cuh"
#include "cuda_helper/hip/hip_runtime_api.h"

#include <iostream>
#include <string>

#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__); }

void gpuAssert(hipError_t code, char const *file, int line, bool abort)
{
    if (code != hipSuccess)
    {
        std::cerr << "Error code " << code << ": " << hipGetErrorString(code) << std::endl;
        std::cerr << "File: " << file << std::endl;
        std::cerr << "Line: " << line << std::endl;
        if(abort)
            exit(code);
    }
}

void ioError(std::string info) {
    std::cerr << "Error reading/writting file. Info: " << info << std::endl;
    exit(EXIT_FAILURE);
}
