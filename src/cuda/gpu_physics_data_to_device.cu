#include "gpu_physics_data_to_device.cuh"

#include "gpu_device_globals.cuh"
#include "utils.hpp"
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>

void gpu_physics_to_device::sendMassStoppingPowerRatio()
{
    std::vector<int> HU_starting_values;
    gpu_physics_to_device::sendMassStoppingPowerRatio(HU_starting_values);
}

void gpu_physics_to_device::sendMassStoppingPowerRatio(std::vector<int>& HU_starting_values)
{
    //  read mass stopping power ratio
    std::string file = std::string(INSTALLATION_PATH) + "/src/phys_data/mass_stopping_power_ratio.dat";
    std::cout << "sendMassStoppingPowerRatio: Reading " << file << std::endl;
    std::ifstream stream(file);
    utils::check_fs(stream, file, "to read mass stopping powers.");

    std::string line;
    std::string dummy;
    // Two dummy lines 
    std::getline(stream, line);
    std::getline(stream, line);
    // Get number of materials
    std::getline(stream, line);
    size_t const n_materials = stoi(line);

    HU_starting_values.resize(n_materials);
    std::vector<float> stp_ratios;
    size_t n_energies;
    float minimum_energy;
    float delta_energy;
    
    // Read data
    for (size_t imat = 0; imat < n_materials; imat++)
    {
        // Get number of energies per material
        std::getline(stream, line);
        std::getline(stream, line);
        std::istringstream ss(line);
        ss >> dummy >> HU_starting_values.at(imat) >>
              dummy >> n_energies >> minimum_energy >> delta_energy;
        std::getline(stream, line);

        if (imat == 0)
        {
            stp_ratios.resize(n_energies*n_materials);
        }

        for (size_t i = 0; i < n_energies; i++)
        {
            std::getline(stream, line);
            std::istringstream ss(line);
            ss >> dummy >> stp_ratios.at(i + imat*n_energies);
        }
    }

    //  transfer to GPU
    minimum_energy *= MeV2eV;
    delta_energy *= MeV2eV;
    hipMemcpyToSymbol(HIP_SYMBOL(stp_ratio_min_e), &minimum_energy, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(stp_ratio_delta_e), &delta_energy, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMallocArray(&stp_ratio_array, &stp_ratio_tex.channelDesc, n_energies, n_materials);
    hipMemcpyToArray(stp_ratio_array, 0,0, &stp_ratios[0], sizeof(float)*stp_ratios.size(), hipMemcpyHostToDevice);
    stp_ratio_tex.filterMode = hipFilterModeLinear;
    hipBindTextureToArray(stp_ratio_tex, stp_ratio_array);
}

void gpu_physics_to_device::sendWaterRestrictedSPower()
{
    std::string file = std::string(INSTALLATION_PATH) + "/src/phys_data/nist_stopping_power_water.dat";
    std::cout << "sendWaterRestrictedSPower: Reading " << file << std::endl;
    std::ifstream stream(file);
    utils::check_fs(stream, file, "to read restricted stopping powers.");

    std::string line;
    // Two dummy lines 
    std::getline(stream, line);
    std::getline(stream, line);
    // Get next
    std::getline(stream, line);
    float dummy, min_energy, delta_energy;
    size_t ndata;
    std::istringstream ss(line);
    ss >> dummy >> min_energy >> dummy >> delta_energy >> ndata;

    min_energy   *= MeV2eV;
    delta_energy *= MeV2eV;

    //  read
    std::vector<float> stp_w(ndata);
    std::vector<float> stp_w_b(ndata);
    //  read info
    std::getline(stream, line);
    std::getline(stream, line);

    for(size_t i = 0; i < ndata; i++)
    {
        std::getline(stream, line, '\n');
        ss.clear();
        ss.str(line);
        ss >> dummy >> dummy >> dummy >> stp_w[i] >> stp_w_b[i];
        stp_w[i] *= MeV2eV;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(stp_w_min_e), &min_energy, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(stp_w_delta_e), &delta_energy, sizeof(float), 0, hipMemcpyHostToDevice);

    //  pass to GPU
    hipMallocArray(&stp_w_array, &stp_w_tex.channelDesc, ndata, 1);
    hipMemcpyToArray(stp_w_array, 0, 0, &stp_w[0], sizeof(float)*ndata, hipMemcpyHostToDevice);
    stp_w_tex.filterMode = hipFilterModeLinear;
    hipBindTextureToArray(stp_w_tex, stp_w_array);

    hipMallocArray(&stp_w_b_coeff_array, &stp_w_b_coeff_tex.channelDesc, ndata, 1);
    hipMemcpyToArray(stp_w_b_coeff_array, 0, 0, stp_w_b.data(), sizeof(float)*ndata, hipMemcpyHostToDevice);
    stp_w_b_coeff_tex.filterMode = hipFilterModeLinear;
    hipBindTextureToArray(stp_w_b_coeff_tex, stp_w_b_coeff_array);
}

void freePhysicsMemory()
{
    hipFreeArray(stp_w_array);
    hipUnbindTexture(stp_w_tex);
    hipFreeArray(stp_w_b_coeff_array);
    hipUnbindTexture(stp_w_b_coeff_tex);
    hipFreeArray(stp_ratio_array);
    hipUnbindTexture(stp_ratio_tex);
}
