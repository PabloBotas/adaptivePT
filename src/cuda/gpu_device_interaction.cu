#include "hip/hip_runtime.h"
#include "gpu_device_interaction.cuh"

__device__ float4 xdata[NRAYS];  // x y z wepl (in cm)
__device__ float4 vxdata[NRAYS]; // vx vy vz (normalized) energy (eV)
__device__ short2 ixdata[NRAYS]; // particle metadata (beam and spot ID)

// CT variables
hipArray *dens;
texture<float, 3, hipReadModeElementType> dens_tex;
__device__ __constant__ float3 ctVoxSize;
__device__ __constant__ int3   ctVox;
__device__ __constant__ size_t ctTotalVoxN;
