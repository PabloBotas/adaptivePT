#include "hip/hip_runtime.h"
#include "gpu_ray_kernel.cuh"
#include "gpu_device_globals.cuh"
#include "gpu_ray_class.cuh"
#include "gpu_geometry_tools.cuh"

__global__ void raytrace_plan_kernel(const int num,
                                     const short* spots_per_field,
                                     const float4* orig_endpoints,
                                     float4 *pos_scorer,
                                     float* traces)
{
    const int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < num)
    {
        // float const max_energy_loss = 0.2; // % of pre-step energy
        Ray ray(xdata[id], vxdata[id], ixdata[id]);
        size_t ind = get_endpoints_index(ray.get_beam_id(), ray.get_spot_id(), spots_per_field);
        int4 vox = get_voxel (ray.get_position());

        VoxelUpdater voxUpdater;
        VoxelStepper voxStepper;

        pos_scorer[ind].w = ray.get_energy();

        // ray.print();
        
        while (ray.is_alive() && vox.w != -1)
        {
            if(traces)
                atomicAdd(&traces[vox.w], 1.0f);

            float step_water, step;
            float max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                         vox, voxUpdater, voxStepper);
            getWaterStep(step, step_water, max_step,
                         ray.get_energy(), ray.get_wepl(), vox);
            ray.move(step, step_water);
            if (step == max_step)
                changeVoxel(vox, voxUpdater, voxStepper);
        }

        // Save scorer
        pos_scorer[ind].x = ray.pos.x;
        pos_scorer[ind].y = ray.pos.y;
        pos_scorer[ind].z = ray.pos.z;

        if (orig_endpoints)
        {
            // printf("Converging to endpoint!!!\n");
            const float sample_energy = 160*MeV2eV;
            const float sample_wepl   = 17.82000; // Janni table for sample energy
            float3 plan_endpoint = make_float3(orig_endpoints[id]);
            ray.set_energy(sample_energy); // sample energy
            ray.set_wepl(sample_wepl);     // Janni table for sample energy
            ray.dir *= ahead_or_behind(ray.dir, plan_endpoint, ray.pos);

            while (ray.is_alive() && vox.w != -1)
            {
                float step_water, step;
                float max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                             vox, voxUpdater, voxStepper,
                                             plan_endpoint);
                getWaterStep(step, step_water, max_step,
                             ray.get_energy(), ray.get_wepl(), vox);
                ray.move(step, step_water);
                if (voxUpdater != NONE)
                    changeVoxel(vox, voxUpdater, voxStepper);
                else
                    break;
            }

            pos_scorer[ind].w = sample_energy - ray.get_energy();
        }

        // ray.print();
    }
}

__device__ void getWaterStep(float& step,
                             float& step_water,
                             const float max_step,
                             const float energy_in,
                             const float avail_wepl,
                             const int4& vox)
{
    // Get density
    float const density  = tex3D(dens_tex, vox.z, vox.y, vox.x);
    // Get stp ratio
    float const mass_stp_ratio = massStpRatio(energy_in, vox);

    // Set steps
    step = max_step;
    step_water = mass_stp_ratio*density*max_step;
    // Verify it's not too much
    if (step_water > avail_wepl)
    {
        step_water = avail_wepl;
        step = step_water/mass_stp_ratio/density;
    }
}

__device__ float massStpRatio(const float energy, const int4& vox)
//  mass stopping power ratio wrt water for a material
{
    float const energy_index = (energy - stp_ratio_min_e)/stp_ratio_delta_e + 0.5;
    int const material_id = tex3D(matid_tex, vox.z, vox.y, vox.x);
    int const material_index = material_id + 0.5;
    return tex2D(stp_ratio_tex, energy_index , material_index);
}


__device__ size_t get_endpoints_index(const short beam_id,
                                      const short spot_id,
                                      const short* spots_per_field)
{   
    unsigned int index = spot_id;
    for (short ibeam = 0; ibeam < beam_id; ibeam++)
    {
        index += spots_per_field[ibeam];
    }
    return index;
}
