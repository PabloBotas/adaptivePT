#include "hip/hip_runtime.h"
#include "gpu_ray_kernel.cuh"

#include "gpu_device_globals.cuh"
#include "gpu_geometry_tools.cuh"
#include "gpu_physics.cuh"
#include "gpu_ray_class.cuh"


__global__ void raytrace_plan_kernel(const short num,
                                     const short* spots_per_field,
                                     const double4* const orig_endpoints,
                                     double4 *pos_scorer,
                                     float* traces)
{
    const int thread = blockIdx.x*blockDim.x + threadIdx.x;
    if(thread < num)
    {
        Ray ray(xdata[thread], vxdata[thread], ixdata[thread]);
        size_t const ind = get_endpoints_index(ray.get_beam_id(),
                                               ray.get_spot_id(),
                                               spots_per_field);
        ray.set_wepl(0);
        int4 vox = get_voxel (ray.get_position());

        VoxelUpdater voxUpdater;
        VoxelStepper voxStepper;
        const double initial_energy = ray.get_energy();
        pos_scorer[ind].w = initial_energy;

        // ray.print();
        while (ray.is_alive() && vox.w >= -1)
        {
            double step_water = 0, step = 0, de = 0;
            double max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                          vox, voxUpdater, voxStepper);
            get_step(step, step_water, de, max_step, ray.get_energy(), vox);
            ray.move(step, step_water, de);

            if(traces)
                score_traces(traces, vox.w, !ray.is_alive());

            if (step == max_step)
                changeVoxel(vox, voxUpdater, voxStepper);
        }

        // Save scorer
        pos_scorer[ind].x = ray.pos.x;
        pos_scorer[ind].y = ray.pos.y;
        pos_scorer[ind].z = ray.pos.z;

        if(orig_endpoints)
        {
            const double accu_wepl = ray.get_wepl();
            const double sample_energy = initial_energy;
            const double sample_wepl   = 0;
                    
            double3 plan_endpoint = make_double3(orig_endpoints[thread]);
            ray.set_energy(sample_energy);
            ray.set_wepl(sample_wepl);
            int sign = ahead_or_behind(ray.dir, plan_endpoint, ray.pos);
            ray.dir *= sign;

            while (ray.is_alive() && vox.w != -1)
            {
                double step_water = 0, step = 0, de = 0;
                double max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                              vox, voxUpdater, voxStepper, plan_endpoint);
                get_step(step, step_water, de, max_step, ray.get_energy(), vox);
                ray.move(step, step_water, de);

                if (voxUpdater != NONE)
                {
                    if (traces)
                    {
                        int tempvox = sign < 0 ? -vox.w : vox.w;
                        score_traces(traces, tempvox, false);
                    }
                    changeVoxel(vox, voxUpdater, voxStepper);
                }
                else
                {
                    if (traces)
                        score_traces(traces, vox.w, true);
                    break;
                }
            }

            const double total_wepl = accu_wepl + sign*ray.get_wepl();
            const double alpha1 = 1.63455120e-05;
            const double alpha2 = 7.72957942e-04;
            const double alpha3 = 3.07077098e-14;
            const double p1 = 1.80084932;
            const double p2 = 0.669078092;
            const double p3 = 1.80100517;
            const double E = pow(total_wepl/alpha1, 1/p1) + pow(total_wepl/alpha2, 1/p2) + pow(total_wepl/alpha3, 1/p3);
            const double delta_E = E - initial_energy;
            pos_scorer[ind].w = delta_E;
        }
    }
}


__device__ void score_traces(float *traces, int voxnum, bool last)
{
    bool del_content = voxnum < 0;
    if (del_content)
    {
        atomicExch(&traces[abs(voxnum)], 0.0f);
    }
    else
    {
        float val = last ? 50.0f : 1.0f;
        atomicExch(&traces[voxnum], val);
    }
}


__device__ size_t get_endpoints_index(const short beam_id,
                                      const short spot_id,
                                      const short* spots_per_field)
{   
    size_t accu_spots = 0;
    for (short i = 0; i < beam_id; i++)
        accu_spots += spots_per_field[i];
    size_t index = accu_spots + spot_id;
    return index;
}
