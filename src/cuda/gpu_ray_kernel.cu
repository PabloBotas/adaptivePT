#include "hip/hip_runtime.h"
#include "gpu_ray_kernel.cuh"

#include "gpu_device_globals.cuh"
#include "gpu_geometry_tools.cuh"
#include "gpu_physics.cuh"
#include "gpu_ray_class.cuh"

__global__ void raytrace_plan_kernel(const int num,
                                     const short* spots_per_field,
                                     const float4* orig_endpoints,
                                     float4 *pos_scorer,
                                     float* traces)
{
    const int id = blockIdx.x*blockDim.x + threadIdx.x;
    if(id < num)
    {
        // float const max_energy_loss = 0.2; // % of pre-step energy
        Ray ray(xdata[id], vxdata[id], ixdata[id]);
        size_t ind = get_endpoints_index(ray.get_beam_id(), ray.get_spot_id(), spots_per_field);
        int4 vox = get_voxel (ray.get_position());

        VoxelUpdater voxUpdater;
        VoxelStepper voxStepper;

        pos_scorer[ind].w = ray.get_energy();

        // ray.print();
        while (ray.is_alive() && vox.w != -1)
        {
            printf("Pos-dir: %f %f %f - %f %f %f - ",
                    ray.pos.x, ray.pos.y, ray.pos.z,
                    ray.dir.x, ray.dir.y, ray.dir.z);
            if(traces)
                atomicAdd(&traces[vox.w], 1.0f);

            float step_water, step;
            float max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                         vox, voxUpdater, voxStepper);
            float de = 0;
            get_water_step(step, step_water, de, max_step,
                           ray.get_energy(), vox);
            ray.move(step, step_water, de);
            if (step == max_step && step > 0)
                changeVoxel(vox, voxUpdater, voxStepper);
        }


        // Save scorer
        pos_scorer[ind].x = ray.pos.x;
        pos_scorer[ind].y = ray.pos.y;
        pos_scorer[ind].z = ray.pos.z;

        if (orig_endpoints)
        {
            // printf("Converging to endpoint!!!\n");
            const float sample_energy = 160*MeV2eV;
            const float sample_wepl   = 17.82000; // Janni table for sample energy
            float3 plan_endpoint = make_float3(orig_endpoints[id]);
            ray.set_energy(sample_energy); // sample energy
            ray.set_wepl(sample_wepl);     // Janni table for sample energy
            ray.dir *= ahead_or_behind(ray.dir, plan_endpoint, ray.pos);

            while (ray.is_alive() && vox.w != -1)
            {
                printf("Pos-dir: %f %f %f - %f %f %f - ",
                        ray.pos.x, ray.pos.y, ray.pos.z,
                        ray.dir.x, ray.dir.y, ray.dir.z);                float step_water, step;
                float max_step = to_boundary(ray.get_position(), ray.get_direction(),
                                             vox, voxUpdater, voxStepper,
                                             plan_endpoint);
                float de = 0;
                get_water_step(step, step_water, de, max_step,
                               ray.get_energy(), vox);
                ray.move(step, step_water, de);
                if (voxUpdater != NONE)
                    changeVoxel(vox, voxUpdater, voxStepper);
                else
                    break;
            }

            pos_scorer[ind].w = sample_energy - ray.get_energy();
        }

        // ray.print();
    }
}

__device__ size_t get_endpoints_index(const short beam_id,
                                      const short spot_id,
                                      const short* spots_per_field)
{   
    unsigned int index = spot_id;
    for (short ibeam = 0; ibeam < beam_id; ibeam++)
    {
        index += spots_per_field[ibeam];
    }
    return index;
}
