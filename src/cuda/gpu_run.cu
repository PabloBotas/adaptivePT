#include "hip/hip_runtime.h"
#include "gpu_run.cuh"

#include "gpu_ray_kernel.cuh"
#include "gpu_source_positioning.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_device_globals.cuh"
#include "gpu_utils.cuh"
#include "special_types.hpp"

#include <iostream>
#include <fstream>
#include <numeric>
#include <vector>
#include <sys/stat.h>

void do_raytrace (const std::vector<short>& spots_per_field,
                  double4* positions_scorer,
                  float* traces_scorer,
                  const Array4<double>& orig_endpoints)
{
    // Set up optional target endpoints
    double4* dev_orig_endpoints = NULL;
    if (!orig_endpoints.empty())
        array_to_device<double4, Vector4_t<double> >(dev_orig_endpoints, orig_endpoints.data(), orig_endpoints.size());

    short* spf_gpu = NULL;
    array_to_device<short>(spf_gpu, spots_per_field.data(), spots_per_field.size());
    
    ushort total_spots = std::accumulate(spots_per_field.begin(), spots_per_field.end(), 0);
    std::cout << std::endl;
    std::cout << "Calculating " << total_spots << " rays ..." << std::endl;
    int nblocks = 1 + (total_spots-1)/NTHREAD_PER_BLOCK_RAYS;
    raytrace_plan_kernel<<<nblocks, NTHREAD_PER_BLOCK_RAYS>>>(total_spots,
                                                              spf_gpu,
                                                              dev_orig_endpoints,
                                                              positions_scorer,
                                                              traces_scorer);
    check_kernel_execution(__FILE__, __LINE__);
    hipFree(spf_gpu);
    hipFree(dev_orig_endpoints);
}


void buffers_to_device(const std::vector<double4>& xbuffer,
                       const std::vector<double4>& vxbuffer,
                       const std::vector<short2>& ixbuffer,
                       const bool alloc)
{
    unsigned int num = xbuffer.size();

    // prepare GPU
    size_t bytes1 = sizeof(double4)*num;
    size_t bytes2 = sizeof(short2)*num;
    if (alloc)
    {
        gpuErrchk( hipMalloc((void **) &xdata,  bytes1) );
        gpuErrchk( hipMalloc((void **) &vxdata, bytes1) );
        gpuErrchk( hipMalloc((void **) &ixdata, bytes2) );
    }
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(xdata),  xbuffer.data(),  bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(vxdata), vxbuffer.data(), bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ixdata), ixbuffer.data(), bytes2, 0, hipMemcpyHostToDevice) );
}

