#include "hip/hip_runtime.h"
#include "gpu_run.cuh"

#include "gpu_ray_kernel.cuh"
#include "gpu_ray_positioning_kernel.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_device_globals.cuh"
#include "special_types.hpp"

#include <iostream>
#include <fstream>
#include <vector>
#include <sys/stat.h>

void calculateRays(const std::vector<float4>& xbuffer,
                   const std::vector<float4>& vxbuffer,
                   const std::vector<short2>& ixbuffer,
                   const std::vector<BeamAngles_t>& angles,
                   const short* spots_per_beam,
                   const float3& ct_offsets,
                   float4* endpoints_scorer,
                   float* traces_scorer)
{
    unsigned int total_spots = rays_to_device(xbuffer, vxbuffer, ixbuffer, angles, ct_offsets);

    short* spots_per_beam_gpu;
    gpuErrchk( hipMalloc((void **) &spots_per_beam_gpu, sizeof(short)*angles.size()) );
    gpuErrchk( hipMemcpy(spots_per_beam_gpu, spots_per_beam, sizeof(short)*angles.size(), hipMemcpyHostToDevice) );
    //      simulate a batch of rays
    std::cout << std::endl;
    std::cout << "Calculating " << total_spots << " rays ..." << std::endl;
    int nblocks = 1 + (total_spots-1)/NTHREAD_PER_BLOCK_RAYS;
    calculateRays_kernel<<<nblocks, NTHREAD_PER_BLOCK_RAYS>>>(total_spots, spots_per_beam_gpu, endpoints_scorer, traces_scorer);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
}

unsigned int rays_to_device(const std::vector<float4>& xbuffer,
                            const std::vector<float4>& vxbuffer,
                            const std::vector<short2>& ixbuffer,
                            const std::vector<BeamAngles_t>& angles,
                            const float3& ct_offsets)
{
    unsigned int num = xbuffer.size();

    // prepare GPU
    size_t bytes1 = sizeof(float4)*num;
    size_t bytes2 = sizeof(short2)*num;
    gpuErrchk( hipMalloc((void **) &xdata, bytes1) );
    gpuErrchk( hipMalloc((void **) &vxdata, bytes1) );
    gpuErrchk( hipMalloc((void **) &ixdata, bytes2) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(xdata),  xbuffer.data(), bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(vxdata), vxbuffer.data(), bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ixdata), ixbuffer.data(), bytes2, 0, hipMemcpyHostToDevice) );

    std::vector<float2> temp(angles.size());
    for (size_t i = 0; i < angles.size(); i++)
    {
        temp[i].x = angles.at(i).gantry;
        temp[i].y = angles.at(i).couch;
    }

    float2* angles_gpu;
    gpuErrchk( hipMalloc((void **) &angles_gpu, sizeof(float2)*angles.size()) );
    gpuErrchk( hipMemcpy(angles_gpu, temp.data(), sizeof(float2)*angles.size(), hipMemcpyHostToDevice) );

    int nblocks = 1 + (num-1)/NTHREAD_PER_BLOCK_SOURCE;
    rays_to_delivery_plane<<<nblocks, NTHREAD_PER_BLOCK_SOURCE>>>(num, angles_gpu, ct_offsets);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipFree(angles_gpu);

    return num;
}

void freeCTMemory()
{
    hipFreeArray(dens);
    hipUnbindTexture(dens_tex);
    hipFreeArray(matid);
    hipUnbindTexture(matid_tex);
    hipFreeArray(stp_ratio_array);
    hipUnbindTexture(stp_ratio_tex);
}
