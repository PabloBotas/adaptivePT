#include "hip/hip_runtime.h"
#include "gpu_run.cuh"

#include "gpu_ray_kernel.cuh"
#include "gpu_source_positioning.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_device_globals.cuh"
#include "gpu_utils.cuh"
#include "special_types.hpp"

#include <iostream>
#include <fstream>
#include <numeric>
#include <vector>
#include <sys/stat.h>

void do_raytrace_plan(const std::vector<short> spots_per_field,
                     float4* positions_scorer,
                     float4* directions_scorer,
                     short2* metadata_scorer,
                     float* traces_scorer)
{
    short* spots_per_field_gpu;
    array_to_device<short>(spots_per_field_gpu, spots_per_field.data(), spots_per_field.size());
    size_t total_spots = std::accumulate(spots_per_field.begin(), spots_per_field.end(), 0);
    std::cout << std::endl;
    std::cout << "Calculating " << total_spots << " rays ..." << std::endl;
    int nblocks = 1 + (total_spots-1)/NTHREAD_PER_BLOCK_RAYS;
    raytrace_plan_kernel<<<nblocks, NTHREAD_PER_BLOCK_RAYS>>>(total_spots,
                                                              spots_per_field_gpu,
                                                              positions_scorer,
                                                              directions_scorer,
                                                              metadata_scorer,
                                                              traces_scorer);
    check_kernel_execution(__FILE__, __LINE__);
    hipFree(spots_per_field_gpu);
}

void do_backtrace_endpoints(const std::vector<short> spots_per_field,
                            float4* positions_scorer,
                            float* traces_scorer)
{
    short* spots_per_field_gpu;
    array_to_device<short>(spots_per_field_gpu, spots_per_field.data(), spots_per_field.size());
    size_t total_spots = std::accumulate(spots_per_field.begin(), spots_per_field.end(), 0);
    std::cout << std::endl;
    std::cout << "Backtracing " << total_spots << " rays ..." << std::endl;
    int nblocks = 1 + (total_spots-1)/NTHREAD_PER_BLOCK_RAYS;
    backtrace_endpoints_kernel<<<nblocks, NTHREAD_PER_BLOCK_RAYS>>>(total_spots,
                                                              spots_per_field_gpu,
                                                              positions_scorer,
                                                              traces_scorer);
    check_kernel_execution(__FILE__, __LINE__);
    hipFree(spots_per_field_gpu);
}

void buffers_to_device(const std::vector<float4>& xbuffer,
                       const std::vector<float4>& vxbuffer,
                       const std::vector<short2>& ixbuffer)
{
    unsigned int num = xbuffer.size();

    // prepare GPU
    size_t bytes1 = sizeof(float4)*num;
    size_t bytes2 = sizeof(short2)*num;
    gpuErrchk( hipMalloc((void **) &xdata,  bytes1) );
    gpuErrchk( hipMalloc((void **) &vxdata, bytes1) );
    gpuErrchk( hipMalloc((void **) &ixdata, bytes2) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(xdata),  xbuffer.data(),  bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(vxdata), vxbuffer.data(), bytes1, 0, hipMemcpyHostToDevice) );
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(ixdata), ixbuffer.data(), bytes2, 0, hipMemcpyHostToDevice) );
}

void buffers_to_device(const std::vector< Vector4_t<float> >& xbuffer,
                       const std::vector< Vector4_t<float> >& vxbuffer,
                       const std::vector< Vector2_t<short> >& ixbuffer)
{
    size_t s = xbuffer.size();
    std::vector<float4> a(s);
    std::vector<float4> b(s);
    std::vector<short2> c(s);
    for (size_t i = 0; i < s; i++)
    {
        a[i].x = xbuffer[i].x;
        b[i].x = vxbuffer[i].x;
        c[i].x = ixbuffer[i].x;
        a[i].y = xbuffer[i].y;
        b[i].y = vxbuffer[i].y;
        c[i].y = ixbuffer[i].y;
        a[i].z = xbuffer[i].z;
        b[i].z = vxbuffer[i].z;
        a[i].w = xbuffer[i].w;
        b[i].w = vxbuffer[i].w;
    }
    buffers_to_device(a, b, c);
}

