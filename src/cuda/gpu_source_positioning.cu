#include "hip/hip_runtime.h"
#include "gpu_source_positioning.cuh"

#include "gpu_device_globals.cuh"
#include "gpu_geometry_tools.cuh"
#include "gpu_utils.cuh"

void virtual_src_to_treatment_plane(const unsigned int num,
                                    const std::vector<BeamAngles_t>& angles,
                                    const float3& ct_offsets)
{
    std::vector<float2> temp(angles.size());
    for (size_t i = 0; i < angles.size(); i++)
    {
        temp[i].x = angles.at(i).gantry;
        temp[i].y = angles.at(i).couch;
    }

    float2* angles_gpu;
    array_to_device<float2>(angles_gpu, temp.data(), angles.size());

    int nblocks = 1 + (num-1)/NTHREAD_PER_BLOCK_SOURCE;
    virtual_src_to_treatment_plane_kernel<<<nblocks, NTHREAD_PER_BLOCK_SOURCE>>>(num, angles_gpu, ct_offsets);
    check_kernel_execution(__FILE__, __LINE__);

    hipFree(angles_gpu);
}

__global__ void virtual_src_to_treatment_plane_kernel(const int num,
                                                      const float2* angles,
                                                      const float3 ct_offsets)
//  set source direction
{
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < num)
    {
        float4 pos  = xdata[tid];
        float4 vel  = vxdata[tid];
        short2 meta = ixdata[tid]; // x = beam_id, y = spot_id
        short beamid = meta.x;

        // Adjust to internal coordinates
        pos = ext_to_int_coordinates(pos);
        vel = ext_to_int_coordinates(vel);

        //  rotate location and direction using gantry and couch
        float gantry = angles[beamid].x;
        float couch  = angles[beamid].y;
        pos = rotate(pos, gantry, couch);
        vel = rotate(vel, gantry, couch);

        // Add offsets
        pos.x -= ct_offsets.x;
        pos.y -= ct_offsets.y;
        pos.z -= ct_offsets.z;

        // Initialize them inside the CT
        pos = ray_trace_to_CT_volume(pos, vel);

        xdata[tid]  = pos;
        vxdata[tid] = vel;
    }
}

__global__ void treatment_plane_to_virtual_src_kernel(const int num,
                                                      const float2* angles,
                                                      const float3 ct_offsets)
//  set source direction
{
    const int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if (tid < num)
    {
        float4 pos  = xdata[tid];
        float4 vel  = vxdata[tid];
        short2 meta = ixdata[tid]; // x = beam_id, y = spot_id
        short beamid = meta.x;

        // TODO: Ray trace to outside of CT volume if necessary????

        pos.x += ct_offsets.x;
        pos.y += ct_offsets.y;
        pos.z += ct_offsets.z;

        //  rotate location using gantry and couch
        float gantry = angles[beamid].x;
        float couch  = angles[beamid].y;
        pos = rotate(pos, -gantry, -couch);
        vel = rotate(vel, -gantry, -couch);

        // Adjust to external coordinates
        pos = int_to_ext_coordinates(pos);
        vel = int_to_ext_coordinates(vel);

        xdata[tid]  = pos;
        vxdata[tid] = vel;
    }
}

__device__ float4 ray_trace_to_CT_volume(const float4& p,
                                         const float4& v)
{
    float4 out = p;

    float3 CT_size = ctVox*ctVoxSize;
    if ((p.x > ctVoxSize.x && p.x < CT_size.x) &&
        (p.y > ctVoxSize.y && p.y < CT_size.y) &&
        (p.z > ctVoxSize.z && p.z < CT_size.z))
        return out;

    // 0.1f is to start a fraction of a voxel inside the CT
    // Distances to faces of the CT
    float d_1x = (0.1f*ctVoxSize.x - p.x)/v.x;
    float d_1y = (0.1f*ctVoxSize.y - p.y)/v.y;
    float d_1z = (0.1f*ctVoxSize.z - p.z)/v.z;
    float d_nx = (CT_size.x - 0.1f*ctVoxSize.x - p.x)/v.x;
    float d_ny = (CT_size.y - 0.1f*ctVoxSize.y - p.y)/v.y;
    float d_nz = (CT_size.z - 0.1f*ctVoxSize.z - p.z)/v.z;

    if((d_1x < 0.0f && d_nx < 0.0f) ||
       (d_1y < 0.0f && d_ny < 0.0f) ||
       (d_1z < 0.0f && d_nz < 0.0f))
    {

    }
    else if((d_1x*d_nx <= 0.0f) &&
            (d_1y*d_ny <= 0.0f) &&
            (d_1z*d_nz <= 0.0f))
    {

    }
    else
    {
        float temp = min(d_1x, d_nx);
        float alphaMin = -1.0f;
        alphaMin = max(alphaMin, temp);

        temp = min(d_1y, d_ny);
        alphaMin = max(alphaMin, temp);

        temp = min(d_1z, d_nz);
        alphaMin = max(alphaMin, temp);

        out.x = p.x + v.x*alphaMin;
        out.y = p.y + v.y*alphaMin;
        out.z = p.z + v.z*alphaMin;
    }

    return out;
}

__device__ __host__ float3 ext_to_int_coordinates(float3 a)
{
    return make_float3(-a.y, -a.x, a.z);
}

__device__ __host__ float4 ext_to_int_coordinates(float4 a)
{
    return make_float4(-a.y, -a.x, a.z, a.w);
}

__device__ __host__ float3 int_to_ext_coordinates(float3 a)
{
    return make_float3(-a.y, -a.x, a.z);
}

__device__ __host__ float4 int_to_ext_coordinates(float4 a)
{
    return make_float4(-a.y, -a.x, a.z, a.w);
}

__device__ float4 rotate(const float4& p, const float& gantry, const float& couch)
{
    float c_couch = __cosf(couch);
    float s_couch = __sinf(couch);
    float c_gantry = __cosf(gantry);
    float s_gantry = __sinf(gantry);

    float4 res;
    res.x = p.x*c_couch - s_couch*(p.y*s_gantry + p.z*c_gantry);
    res.y = p.y*c_gantry - p.z*s_gantry;
    res.z = p.x*s_couch + c_couch*(p.y*s_gantry + p.z*c_gantry);
    res.w = p.w;

    return res;
}

