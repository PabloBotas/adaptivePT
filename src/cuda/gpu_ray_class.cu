#include "hip/hip_runtime.h"
#include "gpu_ray_class.cuh"

#include "gpu_device_globals.cuh"
#include "gpu_physics.cuh"
#include "hip/hip_vector_types.h"

// CONSTRUCTORS --------------------------------------------------------------
__device__ Ray::Ray (double4 x_, double4 vx_, short2 ix_)
{
    pos.x  = x_.x;
    pos.y  = x_.y;
    pos.z  = x_.z;
    wepl   = x_.w;
    dir.x  = vx_.x;
    dir.y  = vx_.y;
    dir.z  = vx_.z;
    energy = vx_.w;

    beam_id = ix_.x;
    spot_id = ix_.y;
}

// BASIC GETTERS -------------------------------------------------------------

__device__ double3 Ray::get_position ()
{
    return pos;
}

__device__ double3 Ray::get_direction ()
{
    return dir;
}

__device__ double Ray::get_wepl ()
{
    return wepl;
}

__device__ double Ray::get_energy ()
{
    return energy;
}

__device__ short Ray::get_beam_id ()
{
    return beam_id;
}

__device__ short Ray::get_spot_id ()
{
    return spot_id;
}

// GETTERS -------------------------------------------------------------------

__device__ bool Ray::is_alive()
{
    _alive = energy > 0;
    return _alive;
}

// BASIC SETTERS -------------------------------------------------------------

__device__ void Ray::set_energy(double m)
{
    energy = m;
}

__device__ void Ray::set_wepl(double m)
{
    wepl = m;
}

__device__ void Ray::set_position (double4 d)
{
    set_position(make_double3(d));
}

__device__ void Ray::set_position (double3 d)
{
    pos = d;
}

__device__ void Ray::set_direction (double4 d)
{
    set_direction(make_double3(d));
}

__device__ void Ray::set_direction (double3 d)
{
    dir = normalize(d);
}

// SETTERS -------------------------------------------------------------------

__device__ void Ray::set_direction_to_point (double4 p)
{
    double3 p2 = make_double3(p);
    set_direction_to_point(p2);
}

__device__ void Ray::set_direction_to_point (double3 p)
{
    dir = p - pos;
    double norm = length(dir);
    dir /= norm;
}

// ACTIONS -------------------------------------------------------------------

__device__ void Ray::move (const double& step,
                           const double& step_water,
                           const double& de)
{
    pos += step*dir;
    energy -= de;
    wepl += step_water;
    if (energy <= stp_w_min_e)
        _alive = false;
}

__device__ void Ray::kill()
{
    energy = 0;
    _alive = false;
}

__device__ void Ray::print()
{
    printf("Beam ID:   %d\n"
           "Spot ID:   %d\n"
           "Position:  %f %f %f\n"
           "Direction: %f %f %f\n"
           "Energy:    %f\n"
           "WEPL:      %f\n",
           beam_id, spot_id, pos.x, pos.y, pos.z,
           dir.x, dir.y, dir.z, energy, wepl);
}

