#include "hip/hip_runtime.h"
#include "initialize_rays.cuh"
#include "tramp.hpp"
#include "spot.hpp"

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MeV2eV 1e6

void init_rays(const Patient_Parameters_t& pat,
               std::vector<float4>& xbuffer,
               std::vector<float4>& vxbuffer,
               std::vector<short2>& ixbuffer)
//  initialize particle buffer
{
    std::cout << "Gantry: " << pat.machine << std::endl;
    for (unsigned int ibeam = 0; ibeam < pat.beam_names.size(); ibeam++)
    {
        std::cout << "Loading beam: " << pat.beam_names.at(ibeam) << std::endl;
        std::cout << "    Beam #:     " << ibeam << std::endl;
        std::cout << "    Tramp file: " << pat.tramp_files.at(ibeam) << std::endl;

        Tramp_t src(pat.tramp_files.at(ibeam), pat.machine);
        src.z = - pat.isocenter_to_beam_distance.at(ibeam); // cm

        if(pat.range_shifters[ibeam].exists)
        {
            RangeShifter_Dims_t rs = pat.range_shifters[ibeam];
            std::cout << "    Range shifter thickness:    " << rs.thick << " cm" << std::endl;
            for (size_t i = 0; i < src.wepls.size(); i++)
            {
                src.wepls.at(i) -= rs.wepl;
            }
        }
        std::cout << "    Source Z plane: " << src.z << " cm" << std::endl;

        float2 SAD = make_float2(pat.virtualSAD.a, pat.virtualSAD.b); // cm

        xbuffer.reserve(src.nspots);
        vxbuffer.reserve(src.nspots);
        ixbuffer.reserve(src.nspots);
        for(unsigned int i=0; i < src.nspots; i++)
        { // LOOP OVER SPOTS
            Spot_t& spot = src.spots[i];

            float3 pos  = getTanslatedPosition(src.z, SAD, make_float2(spot.x, spot.y)); // cm
            float3 dCos = getDirection(pos, make_float2(spot.x, spot.y));
            float energy = src.energies_internal.at(i)*MeV2eV; // eV
            float wepl   = src.wepls.at(i);                    // cm

            pos  = adjust_to_internal_coordinates(pos);
            dCos = adjust_to_internal_coordinates(dCos);

            xbuffer.push_back( make_float4(pos.x, pos.y, pos.z, wepl) );
            vxbuffer.push_back( make_float4(dCos.x, dCos.y, dCos.z, energy) );
            ixbuffer.push_back( make_short2(ibeam, i) );
        }
    }
}

float3 adjust_to_internal_coordinates(float3 a)
{
    return make_float3(-a.y, -a.x, a.z);
}

float3 getTanslatedPosition(float z, float2 SAD, float2 spot)
{
    float3 p;
    p.x = ((SAD.x + z) / SAD.x) * spot.x;
    p.y = ((SAD.y + z) / SAD.y) * spot.y;
    p.z = z;
    return p;
}

float3 getDirection(float3 pos, float2 spot)
{
    float3 dCos;
    float a = (spot.x-pos.x)/abs(pos.z);
    float b = (spot.y-pos.y)/abs(pos.z);
    float norm = sqrt(a*a + b*b + 1.f);
    dCos.x = a/norm;
    dCos.y = b/norm;

    float temp = 1.0f - dCos.x*dCos.x - dCos.y*dCos.y;
    if(temp < 0)
    {
        std::cerr << "Something went wrong calculating direction cosines:\n";
        std::cerr << "    Pos  x:  " << pos.x  << "\n";
        std::cerr << "    Pos  y:  " << pos.y  << "\n";
        std::cerr << "    Pos  z:  " << pos.z  << "\n";
        std::cerr << "    spot x:  " << spot.x << "\n";
        std::cerr << "    spot y:  " << spot.y << std::endl;
        exit(EXIT_FAILURE);
    };
    dCos.z = sqrt(temp);
    return dCos;
}

