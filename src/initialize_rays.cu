#include "hip/hip_runtime.h"
#include "initialize_rays.cuh"

#include "tramp.hpp"
#include "gpu_source_positioning.cuh"
#include "spot.hpp"
#include "hip/hip_vector_types.h"

#include <cmath>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define MeV2eV 1e6

void create_virtual_source_buffers(const Patient_Parameters_t& pat,
                              std::vector<float4>& xbuffer,
                              std::vector<float4>& vxbuffer,
                              std::vector<short2>& ixbuffer)
//  initialize particle buffer
{
    std::cout << "Gantry: " << pat.machine << std::endl;
    for (unsigned int ibeam = 0; ibeam < pat.beam_names.size(); ibeam++) {
        std::cout << "Loading beam: " << pat.beam_names.at(ibeam) << std::endl;
        std::cout << "    Beam #:     " << ibeam << std::endl;
        std::cout << "    Tramp file: " << pat.tramp_files.at(ibeam) << std::endl;

        Tramp_t src(pat.tramp_files.at(ibeam), pat.machine);
        src.z = - pat.isocenter_to_beam_distance.at(ibeam); // cm

        if (pat.range_shifters[ibeam].exists) {
            RangeShifter_Dims_t rs = pat.range_shifters[ibeam];
            std::cout << "    Range shifter thickness:    " << rs.thick << " cm" << std::endl;
            for (size_t i = 0; i < src.wepls.size(); i++) {
                src.wepls.at(i) -= rs.wepl;
            }
        }
        std::cout << "    Source Z plane: " << src.z << " cm" << std::endl;

        float2 SAD = make_float2(pat.virtualSAD.a, pat.virtualSAD.b); // cm

        xbuffer.reserve(src.nspots);
        vxbuffer.reserve(src.nspots);
        ixbuffer.reserve(src.nspots);
        for (unsigned int i=0; i < src.nspots; i++) { // LOOP OVER SPOTS
            Spot_t& spot = src.spots[i];
            float3 pos  = iso_to_virtual_src_pos(src.z, SAD, make_float2(spot.x, spot.y)); // cm
            float3 dCos = getDirection(pos, make_float2(spot.x, spot.y));
            float energy = src.energies_internal.at(i)*MeV2eV; // eV
            float wepl   = src.wepls.at(i);                    // cm

            xbuffer.push_back( make_float4(pos.x, pos.y, pos.z, energy) );
            vxbuffer.push_back( make_float4(dCos.x, dCos.y, dCos.z, wepl) );
            ixbuffer.push_back( make_short2(ibeam, i) );
        }
    }
}

void set_treatment_plane_buffers_ct_space (const Patient_Parameters_t& pat,
                                           const Array4<float>& endpoints,
                                           const Array4<float>& init_pos,
                                           std::vector<float4>& xbuffer,
                                           std::vector<float4>& vxbuffer,
                                           std::vector<short2>& ixbuffer)
{
    size_t s = endpoints.size();
    xbuffer.resize(s);
    vxbuffer.resize(s);
    ixbuffer.resize(s);
    for (size_t i = 0; i < s; i++) {
        float3 start = make_float3(init_pos.at(i).x, init_pos.at(i).y, init_pos.at(i).z);
        float3 end   = make_float3(endpoints.at(i).x, endpoints.at(i).y, endpoints.at(i).z);
        float3 dir   = end - start;
        float3 dCos  = dir/length(dir);
        float wepl   = endpoints.at(i).w;
        float energy = init_pos.at(i).w;
        short2 meta  = get_beam_spot_id(i, pat.spots_per_field);

        // int3 nvox   = make_int3(pat.ct.n.x, pat.ct.n.y, pat.ct.n.z);
        // float3 dvox = make_float3(pat.ct.d.x, pat.ct.d.y, pat.ct.d.z);
        // float3 start2 = ray_trace_to_CT_volume(start, dCos, nvox, dvox);

        xbuffer.at(i)  = make_float4(start, energy);
        vxbuffer.at(i) = make_float4(dCos, wepl);
        ixbuffer.at(i) = meta;

        // printf("%d - 0 - %f %f %f - %f %f %f - %f %f %f - %f %f %f - %f %f %f\n", 
        //        i, xbuffer[i].x, xbuffer[i].y, xbuffer[i].z,
        //        vxbuffer[i].x, vxbuffer[i].y, vxbuffer[i].z,
        //        xbuffer[i].x, xbuffer[i].y, xbuffer[i].z,
        //        vxbuffer[i].x, vxbuffer[i].y, vxbuffer[i].z,
        //        end.x, end.y, end.z);
    }
}

float3 iso_to_virtual_src_pos(float z, float2 SAD, float2 spot)
{
    float3 p;
    p.x = ((SAD.x - std::abs(z)) / SAD.x) * spot.x;
    p.y = ((SAD.y - std::abs(z)) / SAD.y) * spot.y;
    p.z = z;
    return p;
}

float2 virtual_src_to_iso_pos(float3 pos, float2 SAD)
{
    float2 spot;
    spot.x = pos.x * SAD.x / (SAD.x - std::abs(pos.z));
    spot.y = pos.y * SAD.y / (SAD.y - std::abs(pos.z));
    return spot;
}

void virtual_src_to_iso_pos(Array4<float>& pos, SAD_t SAD)
{
    for (size_t i = 0; i < pos.size(); i++) {
        pos.at(i).x = pos.at(i).x * SAD.a / (SAD.a - std::abs(pos.at(i).z));
        pos.at(i).y = pos.at(i).y * SAD.b / (SAD.b - std::abs(pos.at(i).z));
    }
}

float2 virtual_src_to_iso_pos(float3 pos, float3 cos)
{
    float2 spot;
    spot.x = pos.x + std::abs(pos.z)*cos.x / std::sqrt(1-cos.x*cos.x);
    spot.y = pos.y + std::abs(pos.z)*cos.y / std::sqrt(1-cos.y*cos.y);
    return spot;
}

float3 getDirection(float3 pos, float2 spot)
{
    float dx = spot.x-pos.x;
    float dy = spot.y-pos.y;
    float3 dCos;
    dCos.x = dx/std::sqrt(dx*dx+pos.z*pos.z);
    dCos.y = dy/std::sqrt(dy*dy+pos.z*pos.z);
    dCos.z = std::sqrt(1.0f - dCos.x*dCos.x - dCos.y*dCos.y);
    return dCos;
}

short2 get_beam_spot_id (size_t num, const std::vector<short>& spots_per_field)
{
    size_t beamid = 0;
    for (; beamid < spots_per_field.size(); beamid++) {
        if (num >= (size_t)spots_per_field.at(beamid))
            num -= (size_t)spots_per_field.at(beamid);
        else
            break;
    }
    size_t spotid = num;
    return make_short2(beamid, spotid);
}
