#include "hip/hip_runtime.h"
#include "gpu_main.cuh"

#include "patient_parameters.hpp"
#include "volume.hpp"
#include "gpu_ct_to_device.cuh"
#include "initialize_rays.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_run.cuh"
#include "utils.hpp"

#include <iostream>
#include <string>
#include <vector>

void initialize_device(hipEvent_t& start, hipEvent_t& stop)
{
    // mark the start total time timer
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Set device
    int device = 0;
    hipSetDevice(device);
    bool verbose = false;
    printDevProp(device, verbose);
}

void stop_device(hipEvent_t& start, hipEvent_t& stop)
{
    // Get timing
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dt_ms;
    hipEventElapsedTime(&dt_ms, start, stop);
    hipDeviceReset();
    hipDeviceReset();

    std::cout << std::endl;
    std::cout << "Tracing time: "  << dt_ms/1000 << " s" << std::endl;
}

std::vector< Vector4_t<float> > gpu_get_beam_endpoints(const Patient_Parameters_t &pat,
                                                      const Patient_Volume_t &ct)
{
    // Run
    std::vector< Vector4_t<float> > endpoints(pat.total_spots);
    runCalculation(pat, ct, endpoints);
    utils::flip_positions_Z(endpoints, pat.ct);

    return endpoints;
}

void runCalculation(const Patient_Parameters_t& pat,
                    const Patient_Volume_t& ct,
                    std::vector< Vector4_t<float> >& endpoints)
{
    // Set geometry in GPU
    gpu_ct_to_device::sendDimensions(ct);
    std::vector<int> HU_indexes = gpu_ct_to_device::sendMassStoppingPowerRatio();
    gpu_ct_to_device::sendDensities(ct);
    gpu_ct_to_device::sendMaterialId(ct, HU_indexes);

    // Create host buffers and initialize rays
    std::vector<float4> xbuffer;
    std::vector<float4> vxbuffer;
    std::vector<short2> ixbuffer;
    init_rays(pat, xbuffer, vxbuffer, ixbuffer);

    // Create scorer array
    float4* endpoints_scorer = NULL;
    gpuErrchk( hipMalloc( (void **) &endpoints_scorer, sizeof(float4)*pat.total_spots) );
    gpuErrchk( hipMemset( (void *) endpoints_scorer, 0, sizeof(float4)*pat.total_spots) );

    // Calculate rays
#if !defined __OUTPUT_SCORER_VOLUME__
    calculateRays(xbuffer, vxbuffer, ixbuffer,
                  pat.angles, pat.spots_per_field.data(),
                  make_float3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z),
                  endpoints_scorer);
#else
    float* traces_scorer = NULL;
    gpuErrchk( hipMalloc( (void **) &traces_scorer, sizeof(float)*pat.ct.total) );
    gpuErrchk( hipMemset( (void *) traces_scorer, 0, sizeof(float)*pat.ct.total) );  
    calculateRays(xbuffer, vxbuffer, ixbuffer,
                  pat.angles, pat.spots_per_field.data(),
                  make_float3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z),
                  endpoints_scorer, traces_scorer);
    Patient_Volume_t traces(pat.ct);
    gpuErrchk( hipMemcpy(&traces.hu[0], traces_scorer, sizeof(float)*traces.nElements, hipMemcpyDeviceToHost) );
    traces.output("output_volume.mha", "mha");
    traces.output("output_volume.raw", "bin");
    gpuErrchk( hipFree(traces_scorer) );
#endif

    std::vector<float4> temp(endpoints.size());
    gpuErrchk( hipMemcpy(&temp[0], endpoints_scorer, sizeof(float4)*pat.total_spots, hipMemcpyDeviceToHost) );
    for (size_t i = 0; i < temp.size(); i++)
    {
        endpoints.at(i).x = temp.at(i).x;
        endpoints.at(i).y = temp.at(i).y;
        endpoints.at(i).z = temp.at(i).z;
        endpoints.at(i).w = temp.at(i).w;
    }

    // Free memory
    gpuErrchk( hipFree(endpoints_scorer) );
    freeCTMemory();
}


void printDevProp(const int device, bool verbose)
{
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    if(verbose)
    {
        std::cout << "Using device #:        " << device << std::endl;
        std::cout << "Name:                  " << devProp.name << std::endl;
        std::cout << "Compute capability:    " << devProp.major << "." << devProp.minor << std::endl;
        std::cout << "Global memory:         " << devProp.totalGlobalMem/1024.0/1024.0 << " MB" << std::endl;
        std::cout << "Shared memory /block:  " << devProp.sharedMemPerBlock/1024.0 << std::endl;
        std::cout << "Registers /block:      " << devProp.regsPerBlock << std::endl;
        std::cout << "Warp size:             " << devProp.warpSize << std::endl;
        std::cout << "Memory pitch:          " << devProp.memPitch << std::endl;
        std::cout << "Threads /block:        " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "Maximum dim of block:  " << devProp.maxThreadsDim[0] << "," << devProp.maxThreadsDim[1] << "," << devProp.maxThreadsDim[2] << std::endl;
        std::cout << "Maximum dim of grid:   " << devProp.maxGridSize[0] << "," << devProp.maxGridSize[1] << "," << devProp.maxGridSize[2] << std::endl;
        std::cout << "Clock rate:            " << devProp.clockRate/1000000.0 << " GHz" << std::endl;
        std::cout << "Total constant memory: " << devProp.totalConstMem/1024.0 << std::endl;
        std::cout << "Texture alignment:     " << devProp.textureAlignment << std::endl;
        std::cout << "Concurrent copy/exec:  " << (devProp.deviceOverlap ? "Yes" : "No") << std::endl;
        std::cout << "Multiprocessors:       " << devProp.multiProcessorCount << std::endl;
        std::cout << "Kernel timeout:        " << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << std::endl;
    }
    else
    {
        std::cout << "Using card (" << device << "): " << devProp.name << std::endl;
    }
}
