#include "hip/hip_runtime.h"
#include "gpu_main.cuh"

#include "gpu_ct_to_device.cuh"
#include "gpu_device_globals.cuh"
#include "gpu_errorcheck.cuh"
#include "gpu_influence_kernel.cuh"
#include "gpu_physics_data_to_device.cuh"
#include "gpu_run.cuh"
#include "gpu_source_positioning.cuh"
#include "gpu_utils.cuh"
#include "initialize_rays.cuh"
#include "patient_parameters.hpp"
#include "tramp.hpp"
#include "utils.hpp"
#include "volume.hpp"

#include <iostream>
#include <string>
#include <vector>

void gpu_raytrace_original (const Patient_Parameters_t& pat,
                            const Volume_t& ct,
                            Array4<double>& endpoints,
                            Array4<double>& initpos_xbuffer_dbg,
                            Array4<double>& initpos,
                            std::string output_file,
                            Array4<double>& influence)
{
    // Set geometry in GPU
    gpu_ct_to_device::sendGeometries(ct);

    // Create host buffers and initialize rays
    std::vector<double4> xbuffer;
    std::vector<double4> vxbuffer;
    std::vector<short2> ixbuffer;
    create_virtual_source_buffers (pat, xbuffer, vxbuffer, ixbuffer);
    buffers_to_device (xbuffer, vxbuffer, ixbuffer, true);
    virtual_src_to_treatment_plane (xbuffer.size(), pat.angles,
                                    make_double3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z));

    gpuErrchk( hipMemcpyFromSymbol(&(initpos[0].x), xdata, sizeof(double4)*xbuffer.size(), 0, hipMemcpyDeviceToHost) );
    // Copy buffer with initial positions and wepl
    for (size_t i = 0; i < xbuffer.size(); i++)
    {
        initpos_xbuffer_dbg.at(i).x = xbuffer[i].x;
        initpos_xbuffer_dbg.at(i).y = xbuffer[i].y;
        initpos_xbuffer_dbg.at(i).z = xbuffer[i].z;
        initpos_xbuffer_dbg.at(i).w = xbuffer[i].w; // wepl
    }

    gpu_raytrace (pat, endpoints, output_file);

    // INFLUENCE
    std::vector<float> inf_cube(ct.nElements);
    std::vector<float> spot_weights;
    for (size_t i = 0; i < pat.tramp_files.size(); ++i)
    {
        Tramp_t tramp(pat.tramp_files.at(i), pat.machine);
        std::vector<float> w = tramp.get_weights();
        spot_weights.reserve(spot_weights.size() + tramp.nspots);
        spot_weights.insert(spot_weights.end(), w.begin(), w.end());
    }
    gpu_calculate_influence (pat.total_spots, endpoints, influence, spot_weights, inf_cube);
#ifdef __INFLUENCE_MATRICES__
    std::cout << "Writting influence_matrix_CT.dat ..." << std::endl;
    std::ofstream fout("influence_matrix_CT.dat", std::ios::out | std::ios::binary);
    for (size_t i = 0; i < influence.size(); ++i)
        fout.write((char*)&influence[i].w, sizeof(double));

    std::cout << "Writting influence_cube_CT.dat ..." << std::endl;
    std::ofstream fout2("influence_cube_CT.dat", std::ios::out | std::ios::binary);
    fout2.write((char*)&inf_cube[0], inf_cube.size()*sizeof(float));
#endif
    // for (size_t i = 0; i < 10; i++)
    // {
    //     std::cerr << influence[i].x << " " << influence[i].y << " " << influence[i].z << " " << influence[i].w << "\n";
    // }
    freeCTMemory();
}

void gpu_raytrace_warped (const Patient_Parameters_t &pat,
                          const Volume_t &ct,
                          const Array4<double>& orig_endpoints,
                          const Array4<double>& init_pos,
                          Array4<double>& endpoints,
                          std::string output_file,
                          Array4<double>& influence)
{
    // Set geometry in GPU
    gpu_ct_to_device::sendGeometries(ct);

    // Create host buffers and initialize rays
    std::vector<double4> xbuffer;
    std::vector<double4> vxbuffer;
    std::vector<short2> ixbuffer;
    create_treatment_plane_buffers (pat, orig_endpoints, init_pos,
                                    xbuffer, vxbuffer, ixbuffer);
    buffers_to_device (xbuffer, vxbuffer, ixbuffer, false);
    correct_offsets (xbuffer.size(), 
        make_double3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z),
        make_double3(pat.original_ct.offset.x, pat.original_ct.offset.y, pat.original_ct.offset.z));
    Array4<double> off_endpoints = offset_endpoints (orig_endpoints, 
                                       make_double3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z),
                                       make_double3(pat.original_ct.offset.x, pat.original_ct.offset.y, pat.original_ct.offset.z));

    gpu_raytrace (pat, endpoints, output_file, off_endpoints);

    std::vector<float> inf_cube(ct.nElements);
    std::vector<float> spot_weights;
    for (size_t i = 0; i < pat.tramp_files.size(); ++i)
    {
        Tramp_t tramp(pat.tramp_files.at(i), pat.machine);
        std::vector<float> w = tramp.get_weights();
        spot_weights.reserve(spot_weights.size() + tramp.nspots);
        spot_weights.insert(spot_weights.end(), w.begin(), w.end());
    }
    gpu_calculate_influence (pat.total_spots, endpoints, influence, spot_weights, inf_cube);
#ifdef __INFLUENCE_MATRICES__
    std::cout << "Writting influence_CBCT.dat ..." << std::endl;
    std::ofstream fout("influence_matrix_CBCT.dat", std::ios::out | std::ios::binary);
    for (size_t i = 0; i < influence.size(); ++i)
        fout.write((char*)&influence[i].w, sizeof(double));
    
    std::cout << "Writting influence_cube_CBCT.dat ..." << std::endl;
    std::ofstream fout2("influence_cube_CBCT.dat", std::ios::out | std::ios::binary);
    fout2.write((char*)&inf_cube[0], inf_cube.size()*sizeof(float));
#endif
    // for (size_t i = 0; i < influence.size(); i+=6421)
    // {
    //     std::cout << influence[i].x << " " << influence[i].y << " " << influence[i].z << " " << influence[i].w << "\n";
    // }
    freeCTMemory();
}

void gpu_raytrace (const Patient_Parameters_t& pat,
                   Array4<double>& endpoints,
                   std::string output_file,
                   const Array4<double>& orig_endpoints)
{
    // Create scorer array
    double4* pos_scorer = NULL;
    allocate_scorer<double4>(pos_scorer, pat.total_spots);

    // Calculate rays
    if (output_file.empty())
    {
        do_raytrace(pat.spots_per_field, pos_scorer, NULL, orig_endpoints);
    }
    else
    {
        float* traces_scorer = NULL;
        allocate_scorer<float>(traces_scorer, pat.ct.total);
        do_raytrace(pat.spots_per_field, pos_scorer, traces_scorer, orig_endpoints);
        bool long_data = false;
        Volume_t traces(pat.ct, long_data);
        retrieve_scorer<float, float>(&traces.data[0], traces_scorer, traces.nElements);
        // traces.output("output_volume.mha");
        traces.output(output_file);
        gpuErrchk( hipFree(traces_scorer) );
    }

    retrieve_scorer<double, double4>(&(endpoints[0].x), pos_scorer, pat.total_spots);
    // Free memory
    gpuErrchk( hipFree(pos_scorer) );
}

void gpu_calculate_influence (const short& nspots,
                              const Array4<double>& endpoints,
                              Array4<double>& influence,
                              std::vector<float>& spot_weights,
                              std::vector<float>& inf_volume)
{
    // Copy data to device
    double4 *dev_endpoints = NULL;
    array_to_device<double4, Vector4_t<double> >(dev_endpoints, endpoints.data(), endpoints.size());
    // Create scorer array
    double4 *dev_influence = NULL;
    allocate_scorer<double4>(dev_influence, nspots*nspots);
    // Create scorer array
    float *dev_inf_volume = NULL;
    allocate_scorer<float>(dev_inf_volume, inf_volume.size());
    // Create weights array
    float *dev_spot_weights = NULL;
    array_to_device<float>(dev_spot_weights, &spot_weights[0], spot_weights.size());
    
    // Launch influence kernel
    std::cout << "Calculating influence matrix ..." << std::endl;

    int nblocks = 1 + (nspots*nspots-1)/NTHREAD_PER_BLOCK_INFLUENCE;
    get_influence_kernel<<<nblocks, NTHREAD_PER_BLOCK_INFLUENCE>>>(nspots,
                                                                   dev_endpoints,
                                                                   dev_influence,
                                                                   dev_spot_weights,
                                                                   dev_inf_volume);
    check_kernel_execution(__FILE__, __LINE__);
    retrieve_scorer<double, double4>(&(influence[0].x), dev_influence, nspots*nspots);
    retrieve_scorer<float, float>(&(inf_volume[0]), dev_inf_volume, inf_volume.size());
    retrieve_scorer<float, float>(&(spot_weights[0]), dev_spot_weights, spot_weights.size());
    // Free memory
    gpuErrchk( hipFree(dev_endpoints) );
    gpuErrchk( hipFree(dev_influence) );
    gpuErrchk( hipFree(dev_inf_volume) );
    gpuErrchk( hipFree(dev_spot_weights) );
    freeCTMemory();
}

void initialize_device(hipEvent_t& start)
{
    // mark the start total time timer
    hipEventCreate(&start);
    hipEventRecord(start);

    // Set device
    int device = 0;
    hipSetDevice(device);
    bool verbose = false;
    printDevProp(device, verbose);

    gpu_physics_to_device::sendWaterRestrictedSPower();
    gpu_physics_to_device::sendMassStoppingPowerRatio();
    gpu_physics_to_device::sendBraggPeakFits();
}

void stop_device(hipEvent_t& start)
{
    freePhysicsMemory();
    
    // Get timing
    hipEvent_t stop;
    hipEventCreate(&stop);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dt_ms;
    hipEventElapsedTime(&dt_ms, start, stop);
    hipDeviceReset();
    hipDeviceReset();

    std::cout << std::endl;
    std::cout << "Tracing time: "  << dt_ms/1000 << " s" << std::endl;
}

void printDevProp(const int device, bool verbose)
{
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    if(verbose)
    {
        std::cout << "Using device #:        " << device << std::endl;
        std::cout << "Name:                  " << devProp.name << std::endl;
        std::cout << "Compute capability:    " << devProp.major << "." << devProp.minor << std::endl;
        std::cout << "Global memory:         " << devProp.totalGlobalMem/1024.0/1024.0 << " MB" << std::endl;
        std::cout << "Shared memory /block:  " << devProp.sharedMemPerBlock/1024.0 << std::endl;
        std::cout << "Registers /block:      " << devProp.regsPerBlock << std::endl;
        std::cout << "Warp size:             " << devProp.warpSize << std::endl;
        std::cout << "Memory pitch:          " << devProp.memPitch << std::endl;
        std::cout << "Threads /block:        " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "Maximum dim of block:  " << devProp.maxThreadsDim[0] << "," << devProp.maxThreadsDim[1] << "," << devProp.maxThreadsDim[2] << std::endl;
        std::cout << "Maximum dim of grid:   " << devProp.maxGridSize[0] << "," << devProp.maxGridSize[1] << "," << devProp.maxGridSize[2] << std::endl;
        std::cout << "Clock rate:            " << devProp.clockRate/1000000.0 << " GHz" << std::endl;
        std::cout << "Total constant memory: " << devProp.totalConstMem/1024.0 << std::endl;
        std::cout << "Texture alignment:     " << devProp.textureAlignment << std::endl;
        std::cout << "Concurrent copy/exec:  " << (devProp.deviceOverlap ? "Yes" : "No") << std::endl;
        std::cout << "Multiprocessors:       " << devProp.multiProcessorCount << std::endl;
        std::cout << "Kernel timeout:        " << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << std::endl;
    }
    else
    {
        std::cout << "Using card (" << device << "): " << devProp.name << std::endl;
    }
}
