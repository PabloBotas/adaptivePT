#include "hip/hip_runtime.h"
#include "gpu_main.cuh"

#include "patient_parameters.hpp"
#include "gpu_errorcheck.cuh"
#include "gpu_run.cuh"
#include "gpu_geometry_operations.cuh"
#include "gpu_main.cuh"

#include "tramp.hpp"
#include "gpu_run.cuh"
#include "gpu_device_interaction.cuh"
#include "gpu_ct_to_device.cuh"
#include "volume.hpp"

#include <iostream>
#include <string>
#include <vector>


void gpu_launch(const Patient_Parameters_t &pat, const Patient_Volume_t &ct)
{
    int device = 0;
    hipSetDevice(device);
    printDevProp(device, false);

    runCalculation(pat, ct);
}

void runCalculation(const Patient_Parameters_t &pat, const Patient_Volume_t &ct)
{
    // mark the start total time timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    gpu_ct_to_device::setDimensions(ct);
    gpu_ct_to_device::setDensities(ct);

    // the simulation is initialized once, but the calculation is launched nbeams_h times
    for(size_t i=0; i < pat.nbeams; i++)
    {
        // Create tramp object
        Tramp_t tramp(pat.tramp_files.at(i));
        // Create scorer array
        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(nspots), &tramp.nspots, sizeof(unsigned int), 0, hipMemcpyHostToDevice) );
        gpuErrchk( hipMalloc( (void **) &scorer, sizeof(float3)*tramp.nspots) );

        std::vector<float4> xbuffer;
        std::vector<float4> vxbuffer;

        init_rays(pat, i, xbuffer, vxbuffer);
        float3 ct_offsets = make_float3(pat.ct.offset.x, pat.ct.offset.y, pat.ct.offset.z);
        calculateRays(xbuffer, vxbuffer, pat.angles.at(i), ct_offsets);
        // outputScorerResults(i);
        clearScorer(scorer, sizeof(float3)*tramp.nspots);
        std::cout << std::endl;
    }

    // Finalize the entire computation
    freeMemory();

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float dt_ms;
    hipEventElapsedTime(&dt_ms, start, stop);
    hipDeviceReset();
    hipDeviceReset();

    std::cout << std::endl;
    std::cout << "Program time:  "<< dt_ms << "  (ms)" << std::endl;
    std::cout << "Have a nice day!" << std::endl;
}


void printDevProp(const int device, bool verbose)
{
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, device);
    if(verbose)
    {
        std::cout << "Using device #:        " << device << std::endl;
        std::cout << "Name:                  " << devProp.name << std::endl;
        std::cout << "Compute capability:    " << devProp.major << "." << devProp.minor << std::endl;
        std::cout << "Global memory:         " << devProp.totalGlobalMem/1024.0/1024.0 << " MB" << std::endl;
        std::cout << "Shared memory /block:  " << devProp.sharedMemPerBlock/1024.0 << std::endl;
        std::cout << "Registers /block:      " << devProp.regsPerBlock << std::endl;
        std::cout << "Warp size:             " << devProp.warpSize << std::endl;
        std::cout << "Memory pitch:          " << devProp.memPitch << std::endl;
        std::cout << "Threads /block:        " << devProp.maxThreadsPerBlock << std::endl;
        std::cout << "Maximum dim of block:  " << devProp.maxThreadsDim[0] << "," << devProp.maxThreadsDim[1] << "," << devProp.maxThreadsDim[2] << std::endl;
        std::cout << "Maximum dim of grid:   " << devProp.maxGridSize[0] << "," << devProp.maxGridSize[1] << "," << devProp.maxGridSize[2] << std::endl;
        std::cout << "Clock rate:            " << devProp.clockRate/1000000.0 << " GHz" << std::endl;
        std::cout << "Total constant memory: " << devProp.totalConstMem/1024.0 << std::endl;
        std::cout << "Texture alignment:     " << devProp.textureAlignment << std::endl;
        std::cout << "Concurrent copy/exec:  " << (devProp.deviceOverlap ? "Yes" : "No") << std::endl;
        std::cout << "Multiprocessors:       " << devProp.multiProcessorCount << std::endl;
        std::cout << "Kernel timeout:        " << (devProp.kernelExecTimeoutEnabled ? "Yes" : "No") << std::endl;
    }
    else
    {
        std::cout << "Using card (" << device << "): " << devProp.name << std::endl;
    }
}
